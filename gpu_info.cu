#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    // Get the number of CUDA devices
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    if (deviceCount == 0) {
        std::cout << "No CUDA devices found!" << std::endl;
        return 1;
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProps;
        
        // Get properties of the CUDA device
        hipGetDeviceProperties(&deviceProps, i);

        // Print GPU name
        std::cout << "GPU " << i << ": " << deviceProps.name << std::endl;

        // Print CUDA compute capability (version)
        std::cout << "CUDA Compute Capability: " 
                  << deviceProps.major << "." << deviceProps.minor << std::endl;
    }

    return 0;
}
